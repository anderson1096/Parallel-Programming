#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
//incluyendo opencv, para el manejo de imagenes.

#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

//deifinicion global, ya que el opencv no maneja RGB, sino BGR (o sea al reves).
#define RED 2
#define GREEN 1
#define BLUE 0

//la imagen de salida no necesita los canales, ya que al ser en escala de grises, solo tiene un canal.
__host__
void imageToGray(unsigned char *imgInput, int width,int height, unsigned char *imgOutput){
    int row, col;
    for(row = 0; row < height; ++row){
      for(col = 0; col < width; ++col){
        imgOutput[row * width + col] = imgInput[(row * width + col)* 3 + RED]*0.299 + imgInput[(row * width + col)* 3 + GREEN]*0.587 + imgInput[(row * width + col)* 3 + BLUE]*0.114 ;
      }
    }

}

__global__
void imageToGrayGPU(unsigned char *imgInput, int width,int height, unsigned char *imgOutput){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    if(row < height && col < width)){
        imgOutput[row * width + col] = imgInput[(row * width + col)* 3 + RED]*0.299 + imgInput[(row * width + col)* 3 + GREEN]*0.587 + imgInput[(row * width + col)* 3 + BLUE]*0.114 ;
      }
    }

}

int main(int argc, char **argv){
  //defino mis imagenes
  hipError_t error = hipSuccess; // Para controlar errores
  unsigned char *h_imgInicial,*d_imgInicial, *h_imgGray,*d_imgGray, *h_imgGrises;
  char* imageName = argv[1];
  Mat img;

  img = imread(imageName,1);
  if(argc != 2 || img.empty()){
    printf("Error : Imagen no cargada\n");
    return -1;
  }

  //sacamos los atributos de la Imagen

  Size s = img.size();

  int width = s.width;
  int height = s.height;
  int sz = sizeof(unsigned char)*width*height*img.channels();
  int size = sizeof(unsigned char)*width*height;  //no multiplicamos, porque la imagen en escala de grises no tiene canales

  //Separando memoria para la imagen en el device y en la CPU

  h_imgInicial = (unsigned char *) malloc(sz);
  error = hipMalloc((void**)&d_imgInicial,sz);
  if(error != hipSuccess){
  	printf("Error  reservando memoria para d_imgInicial\n");
  	exit(-1);
  }

  //Pasando los datos de la imagen Leída

  h_imgInicial = img.data;
  //copiar los datos de la CPU al Device
  error = hipMemcpy(d_imgInicial,h_imgInicial,sz,hipMemcpyHostToDevice);
  if(error != hipSuccess){
  	printf("Error copiando los datos de h_imgInicial a d_imgInicial\n");
  	exit(-1);
  }

  //separando memoria para las imagenes en grises en CPU y device

  imgGray = (unsigned char *) malloc(size);
  error = hipMalloc((void**)&d_imgGray,size);
  if(error != hipSuccess){
  	printf("Error  reservando memoria para d_imgGray\n");
  	exit(-1);
  } 

  //creamos las dimensiones de la malla para realizar la conversion a grises en la GPU
  dim3  dimBlock(32,32,1); //creando una dimension de 32 bloques, cada bloque con 32 hilos, 1024 hilos en total
  dim3  dimGrid(ceil(width/float(32)),ceil(height/float(32)),1);
  imageToGrayGPU <<dimGrid,dimBlock>>(d_imgInicial, width, height, d_imgGray);


  //Copiamos los resultados de la GPU en la CPU
  error = hipMemcpy(h_imgGrises,d_imgGray,size,hipMemcpyDeviceToHost);
  if(error != hipSuccess){
  	printf("Error copiando de d_imgGray a h_imgGrises\n");
  	exit(-1);
  }

  //imageToGray(imgInicial,width,height,imgGray);

  Mat resultado_gray_imageCPU;
    resultado_gray_imageCPU.create(height,width,CV_8UC1);
    resultado_gray_imageCPU.data = h_imgGray;

   namedWindow("Grises", WINDOW_AUTOSIZE );
   imshow("Grises GPU", resultado_gray_imageCPU);

   waitKey(0);

  	hipFree(d_imgGray); hipFree(d_imgInicial);
   free(h_imgInicial);free(h_imgGray);
   free(h_imgGrises);
  

  return 0;
}

